#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "utils.h"
#include <math.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage, // In array of image
                       unsigned char* const greyImage, //out array of image
                       int numRows, int numCols)
{

    int pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
    int pixel_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (pixel_x >= numRows) {
      return;
    }

    if (pixel_y >= numCols) {
      return;
    }

    uchar4 rgba = rgbaImage[pixel_x * numCols + pixel_y];
    float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
    greyImage[pixel_x * numCols + pixel_y] = channelSum;

}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{

  int threadDim = sqrt(64);
  int numBlocksX = numRows/threadDim+1;
  int numBlocksY = numCols/threadDim+1;

  const dim3 blockSize(threadDim, threadDim, 1);
  const dim3 gridSize(numBlocksX, numBlocksY, 1);

  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}
