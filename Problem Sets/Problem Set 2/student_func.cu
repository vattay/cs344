#include "hip/hip_runtime.h"
// Homework 2
// Image Blurring

#include "utils.h"
#include <stdio.h>

static const int THREAD_DIM = 32;

__global__
void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{

  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows){
    return;
  }

  __shared__ unsigned char sh_channel[THREAD_DIM][THREAD_DIM];
  sh_channel[threadIdx.x][threadIdx.y] = inputChannel[thread_1D_pos];
  __syncthreads();

  float result = 0.f;
  for (int filter_r = -filterWidth/2; filter_r <= filterWidth/2; ++filter_r) {
    for (int filter_c = -filterWidth/2; filter_c <= filterWidth/2; ++filter_c) {
      //Find the global image position for this filter position
      //clamp to boundary of the image
  int image_r = min(max(thread_2D_pos.y + filter_r, 0), static_cast<int>(numRows - 1));
      int image_c = min(max(thread_2D_pos.x + filter_c, 0), static_cast<int>(numCols - 1));

      int local_col = image_c - (blockIdx.x * blockDim.x);
      int local_row = image_r - (blockIdx.y * blockDim.y);

      float image_value;

      if (local_col >= 0 && local_row >=0 && local_col < THREAD_DIM && local_row < THREAD_DIM) {
        image_value = static_cast<float>(sh_channel[local_col][local_row]);
      } else {
        image_value = static_cast<float>(inputChannel[image_r * numCols + image_c]);
      }
      float filter_value = filter[(filter_r + filterWidth/2) * filterWidth + filter_c + filterWidth/2];

      result += image_value * filter_value;
    }
  }

  outputChannel[thread_1D_pos] = result;
  // outputChannel[thread_1D_pos] = inputChannel[thread_1D_pos];
}

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{

  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows){
    return;
  }

  uchar4 rgbaPixel = inputImageRGBA[thread_1D_pos];

  redChannel[thread_1D_pos] = rgbaPixel.x;
  greenChannel[thread_1D_pos] = rgbaPixel.y;
  blueChannel[thread_1D_pos] = rgbaPixel.z;

}

//This kernel takes in three color channels and recombines them
//into one image.  The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  //make sure we don't try and access memory outside the image
  //by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  // checkCudaErrors(hipMemset(d_red, 0, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  // checkCudaErrors(hipMemset(d_green, 0, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));
  // checkCudaErrors(hipMemset(d_blue, 0, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_filter, sizeof(float) * filterWidth * filterWidth));

  //Copy the filter on the host (h_filter) to the memory you just allocated
  //on the GPU.  hipMemcpy(dst, src, numBytes, hipMemcpyHostToDevice);
  //Remember to use checkCudaErrors!
  checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));

}

void your_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redBlurred,
                        unsigned char *d_greenBlurred,
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{
  const dim3 blockSize(THREAD_DIM,THREAD_DIM,1);
  const dim3 gridSize((numCols/THREAD_DIM)+1, (numRows/THREAD_DIM)+1,1);

  //Launch a kernel for separating the RGBA image into different color channels
  separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA, numRows, numCols,
                                            d_red, d_green, d_blue);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  gaussian_blur<<<gridSize, blockSize>>>(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  gaussian_blur<<<gridSize, blockSize>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  gaussian_blur<<<gridSize, blockSize>>>(d_blue, d_blueBlurred, numRows, numCols, d_filter, filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Now we recombine your results. We take care of launching this kernel for you.
  //
  // NOTE: This kernel launch depends on the gridSize and blockSize variables,
  // which you must set yourself.
  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}


//Free all the memory that we allocated
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
  checkCudaErrors(hipFree(d_filter));
}
